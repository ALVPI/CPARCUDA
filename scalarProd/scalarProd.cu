#include "hip/hip_runtime.h"
/* Grupo G11:
	Santiago Gonzalez Rodriguez 
	Alvar Lopez Primo
*/


/*
 * This file contains both device and host code to calculate the
 * scalar product of two vectors of N elements.
 * 
 */

#include <stdio.h>

#define N 1024
#define SEGMENT_SIZE 64

///////////////////////////////////////////////////////////////////////////////
//
// Computes the scalar product of two vectors of N elements on GPU.
//
///////////////////////////////////////////////////////////////////////////////
__global__ void scalarProd(float *C, const float *A, const float *B, int nElem) {
	/*Declaration zone:*/
	/*So, for the thread id inside the grid i need:
		- thread_Idx.x = index of the thread inside his block.
		- block_idx.x = index of the block (the block of threads) inside the grid (the number of blocks that we have)
		- blockDim.x = the number of threads inside the block
		-*/
	int threadIdGrid = thread_Idx.x + block_idx.x * blockDim.x;
	/*The shared memory btw threads that we are going to use to store the partial results*/
	extern __shared__ float sharedMem[];
	/* We need to store that partial results in this variable*/
	float thread_sum_result = 0.0f;
	/*Calculations*/
	if(threadIdGrid < nElem)
	{
		/**6 */
		thread_sum_result = A[threadIdGrid] * B [threadIdGrid];
	}

}

/////////////////////////////////////////////////////////////////
//
// Computes a standard parallel reduction on GPU.
//
/////////////////////////////////////////////////////////////////
__global__ void vectorReduce(float *R, const float *C, int nElem)
{
	// Array in Shared Memory
    extern __shared__ float sdata[];
	
	// COMPLETAR...
}

// -----------------------------------------------
// Host Utility Routines
// -----------------------------------------------
float scalarProd_CPU(float *A, float *B, int nElem)
{
	float suma = 0.0f;	
	for (int i = 0; i < nElem; i++) {
		suma += A[i] * B[i];
	}
	return suma;
}

float randFloat(float low, float high) {
	float t = (float) rand() / (float) RAND_MAX;
	return (1.0f - t) * low + (t * high);
}

// ------------
// Main Program
// ------------
int main( void ) {

	// Array Elements
	int n_elem = N;
	
	// Block Dimension
	int block_dim = SEGMENT_SIZE;
	
	// Number of Blocks
	//int n_block = ( n_elem % block_dim == 0 ) // COMPLETAR...
	int n_block = (n_elem + block_dim -1) / block_dim;
	
	// Execution Configuration Parameters
	dim3 blocks (n_block, n_block);
	dim3 threads(block_dim, block_dim);
	
	// Size (in bytes) Required to Store the Matrix
	size_t n_bytes = (n_elem * sizeof(float));
	
	// Allocate Host Memory
	float *h_A = (float *) malloc( &h_A, n_bytes );
	float *h_B = (float *) malloc( &h_B, n_bytes );
	float *h_R = (float *) malloc( &h_R, n_bytes );
		
	// Initialize Host Data
	srand(123);
	
	// Generating input data on CPU
	for (int i=0; i < n_elem; i++) {
		h_A[i] = randFloat(0.0f, 1.0f);
		h_B[i] = randFloat(0.0f, 1.0f);
	}
	
	// Compute Reference CPU Solution
	float result_cpu = scalarProd_CPU(h_A, h_B, n_elem);
	
	// CUDA Events
	hipEvent_t start, stop;
	
	// Allocate Device Memory
	/*Here im going to explain wich is the function of this pointers on the code*/
	/*d_A the direction where the device will store the first matrix
	Same for the other ones
	d_C is used for store the partial result
	d_R is used for strore the final result*/
	float *d_A, *d_B, *d_C, *d_R;

	/*We just have to have the enougth mem to store the elements of the matrix result */
	hipMalloc((void **)&d_A, sizeof(float)*n_elem);
	hipMalloc((void **)&d_B, sizeof(float)*n_elem);
	hipMalloc((void **)&d_C, sizeof(float)*n_elem);
	hipMalloc((void **)&d_R, sizeof(float)*n_elem);
	
	// Init Events
	hipEventCreate(&start);
	hipEventCreate(&stop );
	
	// Start Time Measurement
    hipEventRecord(start, 0);
	
	// Copy Host Data to Device
	/*Here i have to use cudaMemCpy->hipMemcpy(destination, source, size, cudaMemcpyDirection);*/
	/*So here i just cpy the matrix into the graphic and we dont have to syncronyze bc cpy is a blocking op  */
	hipMemcpy(d_A, h_A, n_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, n_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_C, h_C, n_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_R, h_R, n_bytes, hipMemcpyHostToDevice);

	/*Here we invoke the function for calculate de scalarProduct
	scalarProd<<<numberOfBlocks, Block Dimension>>>(RESULT,MATRIX1, MATRIXB, numberOfElements); */
	scalarProd<<<n_block, block_dim>>>(d_C, d_A, d_B, n_elem);
	hipDeviceSynchronize();
	/*vectorReduce<<<NumberOfBlock, block_dim, size of the shared memory)>>>(Result, source, n_element);*/
	vectorReduce<<<1, block_dim, block_dim*sizeof(float) >>>(d_R, d_C, n_elem);
	
	// Copy Device Data to Host
	/*Here i have to use cudaMemCpy->hipMemcpy(destination, source, size, hipMemcpyDeviceToHost);*/
	cuda_memcpy(h_R, d_R, n_bytes, hipMemcpyDeviceToHost)
	
	
	
	// End Time Measurement
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);

	float kernel_time;
	hipEventElapsedTime(&kernel_time, start, stop);
    printf("Execution Time by the GPU: %.2f\n", kernel_time);

	float result_gpu = 0.0f;
	for (int i=0; i < n_block; i++) {
		result_gpu += h_R[i];
	}
	
	// Free Host Memory
	free(h_A); free(h_B); free(h_R);
	
	// Free Device Memory
	hipFree(d_A); hipFree(d_B);
	hipFree(d_C); hipFree(d_R);
	
	// Destroy Events
	hipEventDestroy(start);
	hipEventDestroy(stop);
	
	if (result_cpu != result_cpu) {
		printf("Test Failed!\n");
		exit(EXIT_FAILURE);
	}
	printf("Test Passed\n");
	exit(EXIT_SUCCESS);
}
